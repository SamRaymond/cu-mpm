#include "mpm_data.h"

//------------------------------------------------------------
// Allocate Particle Data on Device
//------------------------------------------------------------
MpmParticleData allocateParticleDataOnDevice(int numParticles)
{
    MpmParticleData p;
    size_t bytes = numParticles * sizeof(float);

    #define ALLOC_AND_CHECK(ptr)  CUDA_CHECK(hipMalloc((void**)&(ptr), bytes))

    ALLOC_AND_CHECK(p.pos_x);
    ALLOC_AND_CHECK(p.pos_y);
    ALLOC_AND_CHECK(p.vel_x);
    ALLOC_AND_CHECK(p.vel_y);
    ALLOC_AND_CHECK(p.mass);
    ALLOC_AND_CHECK(p.mom_x);
    ALLOC_AND_CHECK(p.mom_y);

    ALLOC_AND_CHECK(p.halfSizeX);
    ALLOC_AND_CHECK(p.halfSizeY);

    ALLOC_AND_CHECK(p.stress_xx);
    ALLOC_AND_CHECK(p.stress_yy);
    ALLOC_AND_CHECK(p.stress_xy);
    ALLOC_AND_CHECK(p.strain_xx);
    ALLOC_AND_CHECK(p.strain_yy);
    ALLOC_AND_CHECK(p.strain_xy);
    ALLOC_AND_CHECK(p.grad_vx_x);
    ALLOC_AND_CHECK(p.grad_vx_y);
    ALLOC_AND_CHECK(p.grad_vy_x);
    ALLOC_AND_CHECK(p.grad_vy_y);

    return p;
}

//------------------------------------------------------------
// Free Particle Data
//------------------------------------------------------------
void freeParticleData(MpmParticleData p)
{
    CUDA_CHECK(hipFree(p.pos_x));
    CUDA_CHECK(hipFree(p.pos_y));
    CUDA_CHECK(hipFree(p.vel_x));
    CUDA_CHECK(hipFree(p.vel_y));
    CUDA_CHECK(hipFree(p.mass));
    CUDA_CHECK(hipFree(p.mom_x));
    CUDA_CHECK(hipFree(p.mom_y));
    CUDA_CHECK(hipFree(p.halfSizeX));
    CUDA_CHECK(hipFree(p.halfSizeY));
    CUDA_CHECK(hipFree(p.stress_xx));
    CUDA_CHECK(hipFree(p.stress_yy));
    CUDA_CHECK(hipFree(p.stress_xy));
    CUDA_CHECK(hipFree(p.strain_xx));
    CUDA_CHECK(hipFree(p.strain_yy));
    CUDA_CHECK(hipFree(p.strain_xy));
    CUDA_CHECK(hipFree(p.grad_vx_x));
    CUDA_CHECK(hipFree(p.grad_vx_y));
    CUDA_CHECK(hipFree(p.grad_vy_x));
    CUDA_CHECK(hipFree(p.grad_vy_y));
}

//------------------------------------------------------------
// Allocate Grid Node Data on Device
//------------------------------------------------------------
MpmGridNodeData allocateGridNodeDataOnDevice(int numNodes)
{
    MpmGridNodeData g;
    size_t bytes = numNodes * sizeof(float);

    CUDA_CHECK(hipMalloc((void**)&g.mass,  bytes));
    CUDA_CHECK(hipMalloc((void**)&g.mom_x, bytes));
    CUDA_CHECK(hipMalloc((void**)&g.mom_y, bytes));
    CUDA_CHECK(hipMalloc((void**)&g.vel_x, bytes));
    CUDA_CHECK(hipMalloc((void**)&g.vel_y, bytes));

    return g;
}

//------------------------------------------------------------
// Free Grid Node Data
//------------------------------------------------------------
void freeGridNodeData(MpmGridNodeData g)
{
    CUDA_CHECK(hipFree(g.mass));
    CUDA_CHECK(hipFree(g.mom_x));
    CUDA_CHECK(hipFree(g.mom_y));
    CUDA_CHECK(hipFree(g.vel_x));
    CUDA_CHECK(hipFree(g.vel_y));
}
